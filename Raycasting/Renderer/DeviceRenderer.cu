#include <glad/glad.h>

#include <cuda_gl_interop.h>

#include <exception>
#include <string>

#include "Core/Core.h"
#include "Core/Vector.h"

#include "DeviceRenderer.h"
#include "Shaders.h"

namespace Raycasting
{

using CudaAssert = Assert<hipError_t, hipGetLastError, hipSuccess, hipGetErrorString>;

DeviceRenderer::DeviceRenderer()
{
}

DeviceRenderer::~DeviceRenderer()
{
}

void DeviceRenderer::InitImpl()
{
	Renderer::InitImpl();

	hipSetDevice(0); CudaAssert();

	const size_t arraySize = MaxSphereCount * sizeof(float);

	hipMalloc(&m_DeviceSpheres.PositionX, arraySize);
	hipMalloc(&m_DeviceSpheres.PositionY, arraySize);
	hipMalloc(&m_DeviceSpheres.PositionZ, arraySize);
	hipMalloc(&m_DeviceSpheres.ColorR, arraySize);
	hipMalloc(&m_DeviceSpheres.ColorG, arraySize);
	hipMalloc(&m_DeviceSpheres.ColorB, arraySize);
	hipMalloc(&m_DeviceSpheres.Radius, arraySize);
	hipMalloc(&m_DeviceSpheres.Shininess, arraySize);
	CudaAssert();

	const size_t arraySize2 = MaxLightCount * sizeof(float);

	hipMalloc(&m_DeviceLights.DirectionX, arraySize2);
	hipMalloc(&m_DeviceLights.DirectionY, arraySize2);
	hipMalloc(&m_DeviceLights.DirectionZ, arraySize2);
	hipMalloc(&m_DeviceLights.ColorR, arraySize2);
	hipMalloc(&m_DeviceLights.ColorG, arraySize2);
	hipMalloc(&m_DeviceLights.ColorB, arraySize2);
	hipMalloc(&m_DeviceLights.Diffuse, arraySize2);
	hipMalloc(&m_DeviceLights.Specular, arraySize2);
	CudaAssert();
}

void DeviceRenderer::ShutdownImpl()
{
	hipFree(m_DeviceLights.DirectionX);
	hipFree(m_DeviceLights.DirectionY);
	hipFree(m_DeviceLights.DirectionZ);
	hipFree(m_DeviceLights.ColorR);
	hipFree(m_DeviceLights.ColorG);
	hipFree(m_DeviceLights.ColorB);
	hipFree(m_DeviceLights.Diffuse);
	hipFree(m_DeviceLights.Specular);

	hipFree(m_DeviceSpheres.PositionX);
	hipFree(m_DeviceSpheres.PositionY);
	hipFree(m_DeviceSpheres.PositionZ);
	hipFree(m_DeviceSpheres.ColorR);
	hipFree(m_DeviceSpheres.ColorG);
	hipFree(m_DeviceSpheres.ColorB);
	hipFree(m_DeviceSpheres.Radius);
	hipFree(m_DeviceSpheres.Shininess);

	hipDestroySurfaceObject(m_Surface); CudaAssert();
	hipGraphicsUnmapResources(1, &m_TextureResource); CudaAssert();
	hipGraphicsUnregisterResource(m_TextureResource); CudaAssert();
	m_TextureResource = nullptr;

	hipDeviceReset(); CudaAssert();

	Renderer::ShutdownImpl();
}

void DeviceRenderer::ResizeImpl(uint32_t width, uint32_t height)
{
	bool noresize = m_ViewportWidth == width && m_ViewportHeight == height;

	Renderer::ResizeImpl(width, height);

	if (noresize) return;

	// drop cuda's lock on the buffer
	if (m_TextureResource != nullptr)
	{
		hipDestroySurfaceObject(m_Surface); CudaAssert();
		hipGraphicsUnmapResources(1, &m_TextureResource); CudaAssert();
		hipGraphicsUnregisterResource(m_TextureResource); CudaAssert();
	}

	// resize the buffer with opengl
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, m_ViewportWidth, m_ViewportHeight,
		0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr); GlAssert();

	// lock the buffer again to cuda
	hipGraphicsGLRegisterImage(&m_TextureResource, m_TextureId, GL_TEXTURE_2D,
		hipGraphicsRegisterFlagsSurfaceLoadStore); CudaAssert();
	hipGraphicsMapResources(1, &m_TextureResource); CudaAssert();

	hipArray *array;
	hipGraphicsSubResourceGetMappedArray(&array, m_TextureResource, 0, 0); CudaAssert();

	hipResourceDesc desc;
	memset(&desc, 0, sizeof(hipResourceDesc));
	desc.resType = hipResourceTypeArray;
	desc.res.array.array = array;

	hipCreateSurfaceObject(&m_Surface, &desc);
	CudaAssert();
}

void DeviceRenderer::RenderImpl()
{
	Renderer::RenderImpl();

	{
		Timer timer("Scene Upload");

		const SphereRef &spheresRef = s_ActiveScene->GetSpheres();
		const LightRef &lightsRef = s_ActiveScene->GetLights();

		const size_t arraySize = s_ActiveScene->GetSphereCount() * sizeof(float);
		
		hipMemcpy(m_DeviceSpheres.PositionX, spheresRef.PositionX, arraySize, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceSpheres.PositionY, spheresRef.PositionY, arraySize, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceSpheres.PositionZ, spheresRef.PositionZ, arraySize, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceSpheres.ColorR, spheresRef.ColorR, arraySize, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceSpheres.ColorG, spheresRef.ColorG, arraySize, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceSpheres.ColorB, spheresRef.ColorB, arraySize, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceSpheres.Radius, spheresRef.Radius, arraySize, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceSpheres.Shininess, spheresRef.Shininess, arraySize, hipMemcpyHostToDevice);
		CudaAssert();

		const size_t arraySize2 = s_ActiveScene->GetLightCount() * sizeof(float);

		hipMemcpy(m_DeviceLights.DirectionX, lightsRef.DirectionX, arraySize2, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceLights.DirectionY, lightsRef.DirectionY, arraySize2, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceLights.DirectionZ, lightsRef.DirectionZ, arraySize2, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceLights.ColorR, lightsRef.ColorR, arraySize2, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceLights.ColorG, lightsRef.ColorG, arraySize2, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceLights.ColorB, lightsRef.ColorB, arraySize2, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceLights.Diffuse, lightsRef.Diffuse, arraySize2, hipMemcpyHostToDevice);
		hipMemcpy(m_DeviceLights.Specular, lightsRef.Specular, arraySize2, hipMemcpyHostToDevice);
		CudaAssert();
	}

	Camera &camera = s_ActiveScene->GetCamera();
	mat4<float> view = camera.GetInvViewMatrix();
	mat4<float> proj = camera.GetInvProjectionMatrix();
	vec3<float> position = camera.GetPosition();

	int tx = 8, ty = 8;
	dim3 blocks(m_ViewportWidth / tx + 1, m_ViewportHeight / ty + 1);
	dim3 threads(tx, ty);

	{
		Timer timer("Pixel Shader");
		TraceRays<<<blocks, threads>>>(m_Surface, m_ViewportWidth, m_ViewportHeight, view,  proj, position,
			s_ActiveScene->GetSphereCount(), m_DeviceSpheres.GetRef(), s_ActiveScene->GetLightCount(), m_DeviceLights.GetRef());

		hipDeviceSynchronize();
		CudaAssert();
	}
}

}